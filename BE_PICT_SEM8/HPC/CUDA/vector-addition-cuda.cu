
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
using namespace std;

// blockIdx.x * blockDim.x + threadIdx.x
__global__ void vecAdd(int* A, int* B, int* C, int size) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	if (tid < size)
		C[tid] = A[tid] + B[tid];
}

int main() {

	int size = 5;
	vector<int> A = {1, 2, 3, 4, 5};
	vector<int> B = {2, 3, 4, 5, 6};
	vector<int> C;
	C.resize(size);

	int *X, *Y, *Z;

	int allocSize = size*sizeof(int);

	hipMalloc(&X, allocSize);
	hipMalloc(&Y, allocSize);
	hipMalloc(&Z, allocSize);

	hipMemcpy(X, A.data(), allocSize, hipMemcpyHostToDevice);
	hipMemcpy(Y, B.data(), allocSize, hipMemcpyHostToDevice);

	vecAdd<<<1, 255>>>(X,Y,Z,size);

	hipMemcpy(C.data(), Z, allocSize, hipMemcpyDeviceToHost);

	for(int i = 0; i<size; i++)
		cout << C[i] << "\t";

	cout << endl;

	return 0;
}
